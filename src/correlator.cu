#include "hip/hip_runtime.h"
#include "correlator.cuh"
#include <cassert>
#include <cmath>

#ifdef _DEBUG_BUILD
#define CHECK(call)                                                                       \
    {                                                                                     \
        const hipError_t err = call;                                                     \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            std::cerr << __FILE__ << ":" << __LINE__ <<": ERROR: " << hipGetErrorString(err) << std::endl;\
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

#define CHECK_KERNELCALL()                                                                \
    {                                                                                     \
        const hipError_t err = hipGetLastError();                                       \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            std::cerr << __FILE__ << ":" << __LINE__ <<": ERROR: " << hipGetErrorString(err) << std::endl;\
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

#else
#define CHECK(call) call;                                                                   
#define CHECK_KERNELCALL()                                                         
#endif // _DEBUG_BUILD


// Macros

#define SHARED_OFF(sensor, bin, channel, bin_size, num_sensors_per_block) \
        (sensor) * (bin_size) + (bin) * (bin_size) * (num_sensors_per_block) + (channel)

#define SHARED_OFF_B(sensor, bin, num_sensors_per_block) \
        (sensor) + (bin) * (num_sensors_per_block)

#define GLOBAL_OFF(sensor, bin, channel, bin_size, num_sensors_per_block, num_bins, first_block_sensor) \
        (first_block_sensor) * (num_bins) * (bin_size) + SHARED_OFF(sensor, bin, channel, bin_size, num_sensors_per_block)

#define GLOBAL_OFF_B(sensor, bin, num_sensors_per_block, num_bins, first_block_sensor) \
        (first_block_sensor) * (num_bins) + SHARED_OFF_B(sensor, bin, num_sensors_per_block)


// Kernels

__inline__ __device__ int
MultiTau::insert_until_bin(const int instant, const int num_bins){
    int mask = 1;

    for (int i = 0; i < num_bins; ++i) {
        if ((instant & mask) != 0){
            return i + 1;
        }
        mask = mask << 1;
    }
    return num_bins;
};

template <typename T>
__global__ void 
MultiTau::correlate<T>(T * new_values, const int timepoints, int instants_processed, T * shift_register, int * shift_positions, T * accumulators, T * correlation, const int num_bins, const int num_sensors){

    int num_sensors_per_block = blockDim.y;
    int bin_size = blockDim.x;

    int sensor = threadIdx.y;  // relative sensor id inside the block -> 0..num_sensors_per_block
    int channel = threadIdx.x; // channels goes from 0..bin_size

    int first_block_sensor = blockIdx.x * num_sensors_per_block; // id for the first sensor of the block
    int sensor_gp = first_block_sensor + sensor; // gp stands for global-position, this is the global sensor id

    // Due to templates memory needs to be assigned as unsigned char 
    extern __shared__  unsigned char total_shared_memory[];

    T * block_shift = reinterpret_cast<T *>(&total_shared_memory);
    T * block_correlation = &block_shift[num_sensors_per_block * num_bins * bin_size];
    T * block_accumulators = &block_correlation[num_sensors_per_block * num_bins * bin_size];
    int * block_shift_pos = reinterpret_cast<int *>(&block_accumulators[num_sensors_per_block * num_bins]);

    int clamp_mask = first_block_sensor + num_sensors_per_block >= num_sensors;
    num_sensors_per_block = (num_sensors - first_block_sensor) * clamp_mask + num_sensors_per_block * (1 - clamp_mask);
    

    if ( sensor_gp < num_sensors ) {

        // Copy correlator arrays from global memory to shared memory
        for (int bin = 0; bin < num_bins; ++bin) {
            block_shift[SHARED_OFF(sensor, bin, channel, bin_size, num_sensors_per_block)] =  shift_register[GLOBAL_OFF(sensor, bin, channel, bin_size, num_sensors_per_block, num_bins, first_block_sensor)];
            block_correlation[SHARED_OFF(sensor, bin, channel, bin_size, num_sensors_per_block)] =  correlation[GLOBAL_OFF(sensor, bin, channel, bin_size, num_sensors_per_block, num_bins, first_block_sensor)];

            block_accumulators[SHARED_OFF_B(sensor, bin, num_sensors_per_block)] = accumulators[GLOBAL_OFF_B(sensor, bin, num_sensors_per_block, num_bins, first_block_sensor)];
            block_shift_pos[SHARED_OFF_B(sensor, bin, num_sensors_per_block)] = shift_positions[GLOBAL_OFF_B(sensor, bin, num_sensors_per_block, num_bins, first_block_sensor)];

        }


        __syncthreads();

        // Add new point of the series to the correlator
        for (int instant = 0; instant < timepoints ; ++instant) {
            ++instants_processed;

            int insert_channel_fb = block_shift_pos[SHARED_OFF_B(sensor, 0, num_sensors_per_block)]; // fb stands for first bin
            T new_value = new_values[instant * num_sensors + sensor_gp];
            
            if (channel == 0) { // only one thread add the new_value to the sensor shift register
                block_shift[SHARED_OFF(sensor, 0, insert_channel_fb, bin_size, num_sensors_per_block)] = new_value;
                block_accumulators[SHARED_OFF_B(sensor, 0, num_sensors_per_block)] += new_value;
            }

            __syncthreads();

            block_correlation[SHARED_OFF(sensor, 0, channel, bin_size, num_sensors_per_block)] +=  block_shift[SHARED_OFF(sensor, 0, insert_channel_fb, bin_size, num_sensors_per_block)] * block_shift[SHARED_OFF(sensor, 0, (insert_channel_fb - channel + bin_size) & ( bin_size - 1 ), bin_size, num_sensors_per_block)];
            block_shift_pos[SHARED_OFF_B(sensor, 0, num_sensors_per_block)] = (insert_channel_fb + 1) & (bin_size-1);
            

            size_t max_bin = insert_until_bin(instants_processed, num_bins);
            
            for(int bin = 1; bin < max_bin ; ++bin) {

                int insert_channel = block_shift_pos[SHARED_OFF_B(sensor, bin, num_sensors_per_block)];
                
                if (channel == 0) { // only one thread add the new_value to the sensor shift register
                    block_shift[SHARED_OFF(sensor, bin, insert_channel, bin_size, num_sensors_per_block)] = block_accumulators[SHARED_OFF_B(sensor, bin-1, num_sensors_per_block)];
                    block_accumulators[SHARED_OFF_B(sensor, bin, num_sensors_per_block)] += block_accumulators[SHARED_OFF_B(sensor, bin-1, num_sensors_per_block)];
                }
                __syncthreads();

                block_accumulators[SHARED_OFF_B(sensor, bin-1, num_sensors_per_block)] = 0;

                block_correlation[SHARED_OFF(sensor, bin, channel, bin_size, num_sensors_per_block)] +=  block_shift[SHARED_OFF(sensor, bin, insert_channel, bin_size, num_sensors_per_block)] * block_shift[SHARED_OFF(sensor, bin, (insert_channel - channel + bin_size) & (bin_size -1), bin_size, num_sensors_per_block)] * (channel - bin_size/M >= 0); // only half of the channel needs to be computed, the last member of the multiplication is used to remove garbage from unused channels
                block_shift_pos[SHARED_OFF_B(sensor, bin, num_sensors_per_block)] = (insert_channel + 1) & (bin_size-1);

            }

        }
        
        // Copy correlator arrays from global memory to shared memory
        for (int bin = 0; bin < num_bins; ++bin) {
            shift_register[GLOBAL_OFF(sensor, bin, channel, bin_size, num_sensors_per_block, num_bins, first_block_sensor)] = block_shift[SHARED_OFF(sensor, bin, channel, bin_size, num_sensors_per_block)];
            correlation[GLOBAL_OFF(sensor, bin, channel, bin_size, num_sensors_per_block, num_bins, first_block_sensor)] = block_correlation[SHARED_OFF(sensor, bin, channel, bin_size, num_sensors_per_block)];

            accumulators[GLOBAL_OFF_B(sensor, bin, num_sensors_per_block, num_bins, first_block_sensor)] = block_accumulators[SHARED_OFF_B(sensor, bin, num_sensors_per_block)];
            shift_positions[GLOBAL_OFF_B(sensor, bin, num_sensors_per_block, num_bins, first_block_sensor)] = block_shift_pos[SHARED_OFF_B(sensor, bin, num_sensors_per_block)];

        }

    }
};


template <typename T>
Correlator<T>::Correlator(const int t_num_bins, const int t_bin_size, const int t_num_sensors, const int t_packet_size, const int t_device, const bool t_debug){    
    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);

    hipDeviceProp_t device_properties;
    hipGetDeviceProperties(&device_properties, t_device);
    
    num_bins = t_num_bins;
    bin_size = t_bin_size;
    num_sensors = t_num_sensors;
    packet_size = t_packet_size;
    debug = t_debug;

    max_tau = bin_size * std::pow(2, num_bins);
    num_taus = (((num_bins-1) * (bin_size/M) + bin_size));

    int max_shared_mem_per_block = std::floor((double)device_properties.sharedMemPerMultiprocessor / device_properties.multiProcessorCount); // Used to achieve maximum MP usage

    int accumulators_block_usage = num_bins * sizeof(T);
    int shift_registers_block_usage = (num_bins * bin_size) * sizeof(T);
    int correlations_block_usage = (num_bins * bin_size) * sizeof(T);
    int shift_registers_pos_block_usage = num_bins * sizeof(int);

    shared_memory_per_block = accumulators_block_usage + shift_registers_block_usage + correlations_block_usage + shift_registers_pos_block_usage;

    num_sensors_per_block = std::max(std::floor((double) max_shared_mem_per_block / shared_memory_per_block), (double) 1);

    shared_memory_per_block *= num_sensors_per_block;

    number_of_blocks = dim3(std::ceil((double) num_sensors / num_sensors_per_block), 1 , 1);
    threads_per_block = dim3(bin_size, num_sensors_per_block, 1);

    if (debug){
        std::cout << "[INFO] --------------------------------------" << std::endl;
        std::cout << "[INFO] Number of bins: " << num_bins << std::endl;
        std::cout << "[INFO] Size of bins: " << bin_size << std::endl;
        std::cout << "[INFO] Number of sensors: " << num_sensors << std::endl;
        std::cout << "[INFO] Packet size: " << packet_size << std::endl;
        std::cout << "[INFO] Number of sensors per block: " << num_sensors_per_block << std::endl;
        std::cout << "[INFO] Max tau possible: " << max_tau << std::endl;
        std::cout << "[INFO] Number of taus possible: " << num_taus << std::endl;
        std::cout << "[INFO] --------------------------------------" << std::endl;
        std::cout << "[INFO] Number of blocks: (" << number_of_blocks.x << "," << number_of_blocks.y << "," << number_of_blocks.z << ")" << std::endl;
        std::cout << "[INFO] Threads per blocks: (" << threads_per_block.x << "," << threads_per_block.y << "," << threads_per_block.z << ")" << std::endl;
        std::cout << "[INFO] Shared memory available per multi processor: " << device_properties.sharedMemPerMultiprocessor  << " B" << std::endl;
        std::cout << "[INFO] Number of available multi processors: " << device_properties.multiProcessorCount  << std::endl;
        std::cout << "[INFO] Maximum shared memory available per block: " << max_shared_mem_per_block << " B" << std::endl;
        std::cout << "[INFO] Shared memory used per block: " << shared_memory_per_block << " B" << std::endl;
        std::cout << "[INFO] Shared memory used for shift registers per block: " << shift_registers_block_usage * num_sensors_per_block << " B" << std::endl;
        std::cout << "[INFO] Shared memory used for shift registers positions per block: " << shift_registers_pos_block_usage * num_sensors_per_block << " B" << std::endl;
        std::cout << "[INFO] Shared memory used for correlations per block: " << correlations_block_usage * num_sensors_per_block << " B" << std::endl;
        std::cout << "[INFO] Shared memory used for accumulators per block: " << accumulators_block_usage * num_sensors_per_block << " B" << std::endl;
        std::cout << "[INFO] --------------------------------------" << std::endl;

    }
    
    assert(shared_memory_per_block <= device_properties.sharedMemPerBlock && "ERROR: current configuration exceed device shared memory limits");
    assert(threads_per_block.x * threads_per_block.y < device_properties.maxThreadsPerBlock && "ERROR: current configuration exceed device max num thread per block");

};

template <typename T>
Correlator<T>::~Correlator(){
    // Free host memory
    if (correlation != nullptr){
        free(correlation);
    }
    if (taus != nullptr){
        free(taus);
    }

    // Free device memory
    if (d_shift_register != nullptr){
        CHECK(hipFree(d_shift_register));
    }

    if (d_shift_positions != nullptr){
        CHECK(hipFree(d_shift_positions));
    }

    if (d_accumulators != nullptr){
        CHECK(hipFree(d_accumulators));
    }

    if (d_correlation != nullptr){
        CHECK(hipFree(d_correlation));
    }

    if (d_new_values != nullptr){
        CHECK(hipFree(d_new_values));
    }

    hipDeviceReset();
};

template <typename T>
void Correlator<T>::alloc(){
    if (debug) std::cout << "[INFO] Allocating device arrays into global memory" << std::endl;

    CHECK(hipMalloc(&d_shift_register, num_bins * bin_size * num_sensors * sizeof(T)));
    CHECK(hipMalloc(&d_shift_positions, num_bins * num_sensors * sizeof(int)));

    CHECK(hipMalloc(&d_accumulators, num_bins * num_sensors * sizeof(T)));

    CHECK(hipMalloc(&d_correlation, num_bins * bin_size * num_sensors * sizeof(T)));

    CHECK(hipMalloc(&d_new_values, packet_size * num_sensors * sizeof(T)));

    if (debug) std::cout << "[INFO] Initializing device arrays" << std::endl;

    CHECK(hipMemset(d_shift_register, 0, num_bins * bin_size * num_sensors * sizeof(T)));
    CHECK(hipMemset(d_shift_positions, 0, num_bins * num_sensors * sizeof(int)));

    CHECK(hipMemset(d_accumulators, 0, num_bins * num_sensors * sizeof(T)));

    CHECK(hipMemset(d_correlation, 0 , num_bins * bin_size * num_sensors * sizeof(T)));

    CHECK(hipMemset(d_new_values,0,  packet_size * num_sensors * sizeof(T)));


    if (debug) std::cout << "[INFO] Alocating device arrays" << std::endl;

    correlation = (T*)malloc(num_bins * bin_size * num_sensors * sizeof(T));

};

template <typename T>
void Correlator<T>::correlate(const T * new_values, const int timepoints){

    transfered = false;

    if (debug) std::cout << "[INFO] Allocating and copying new values to gpu array" << std::endl;

    CHECK(hipMemcpy(d_new_values, new_values, timepoints * num_sensors * sizeof(T), hipMemcpyHostToDevice));

    if (debug) std::cout << "[INFO] Starting correlation" << std::endl;

    MultiTau::correlate<T><<<number_of_blocks, threads_per_block, shared_memory_per_block>>>(d_new_values, timepoints, instants_processed, d_shift_register, d_shift_positions, d_accumulators, d_correlation, num_bins, num_sensors);
    hipDeviceSynchronize();
    CHECK_KERNELCALL();
    
    if (debug) std::cout << "[INFO] Instant Processed: " << instants_processed << std::endl;

    instants_processed += timepoints;
};

template <typename T>
void Correlator<T>::transfer(){    

    if (debug) std::cout << "[INFO] Transfering data from device memory to host memory" << std::endl;

    if (!transfered){
        CHECK(hipMemcpy(correlation, d_correlation, num_bins * bin_size * num_sensors * sizeof(T), hipMemcpyDeviceToHost));
        transfered = true;
    }

    if (debug) std::cout << "[INFO] Data transfered" << std::endl;
}


template <typename T>
T Correlator<T>::get(const int sensor, const int lag){
    assert(transfered && "ERROR: Data not transfered from device memory to host memory");

    int block = std::floor((double) sensor / num_sensors_per_block);
    int sensor_rp = sensor - block * num_sensors_per_block;
    int fsb = block * num_sensors_per_block;

    if (lag < bin_size)
        return correlation[block * num_sensors_per_block * num_bins * bin_size + sensor_rp * bin_size + lag];

    int n_spb_adj = (fsb + num_sensors_per_block >= num_sensors) ? (num_sensors - fsb) : num_sensors_per_block;   
    int bin = std::ceil((double)(lag - bin_size + 1) / (double)(bin_size/2));
    int channel = (lag - bin_size) - (bin_size/2) * (bin-1) + (bin_size/2);
    
    return correlation[block * num_sensors_per_block * num_bins * bin_size + sensor_rp * bin_size  +  bin * n_spb_adj * bin_size + channel];
};

template <typename T>
void Correlator<T>::reset(){

    if (debug) std::cout << "[INFO] Resetting all device arrays to zero" << std::endl;

    CHECK(hipMemset(d_shift_register, 0, num_bins * bin_size * num_sensors * sizeof(T)));
    CHECK(hipMemset(d_shift_positions, 0, num_bins * num_sensors * sizeof(int)));

    CHECK(hipMemset(d_accumulators, 0, num_bins * num_sensors * sizeof(T)));

    CHECK(hipMemset(d_correlation, 0 , num_bins * bin_size * num_sensors * sizeof(T)));
    
    CHECK(hipMemset(d_new_values, 0 , packet_size * num_sensors * sizeof(T)));

    if (debug) std::cout << "[INFO] Resetting all host arrays to zero" << std::endl;

    memset(correlation, 0, num_bins * bin_size * num_sensors * sizeof(T));

    instants_processed = 0;
    transfered = false;
};

// Needed for the template
template class Correlator<int16_t>;
template class Correlator<int32_t>;
template class Correlator<int64_t>;

template class Correlator<uint16_t>;
template class Correlator<uint32_t>;
template class Correlator<uint64_t>;

template class Correlator<double>;
template class Correlator<float>;